#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>

#include "kernel.h"

__device__ float tempParticle1[NUM_OF_DIMENSIONS];
__device__ float tempParticle2[NUM_OF_DIMENSIONS];


__device__ float cuda_calculate_F(float x,float y, bool b_[Nb_x][Nb_y], struct point2d r[Nb_x][Nb_y], float phi[Nb_x][Nb_y])
{
  float d = 0;
  for (int i = 0; i < Nb_x; i++)
  {
    for (int j = 0; j < Nb_y; j++)
    {
      if (b_[i][j] == 0) d = d + phi[i][j]*sqrt((x-r[i][j].x)*(x-r[i][j].x) + (y-r[i][j].y)*(y-r[i][j].y));
    }
  } 
  return d;
}

__device__ float cuda_calc_d(float x1,float y1,float x2,float y2)
{
    return sqrt((x1-x2)*(x1-x2) + (y1-y2)*(y1-y2));
}

__device__ bool cuda_check_if_grid_is_inside_R(float x,float y, struct point2d r)
{
    float rx1 = r.x + res/2;
    float rx2 = r.x + res/2;
    float rx3 = r.x - res/2;
    float rx4 = r.x - res/2;

    float ry1 = r.y - res/2;
    float ry2 = r.y + res/2;
    float ry3 = r.y + res/2;
    float ry4 = r.y - res/2;
    if ( (cuda_calc_d(x,y,rx1,ry1) < R) && (cuda_calc_d(x,y,rx2,ry2) < R) && (cuda_calc_d(x,y,rx3,ry3) < R) && (cuda_calc_d(x,y,rx4,ry4) < R) )
    {
        return 1;
    }
    else 
    {
        return 0;
    }
}

__device__ void cuda_forward_euler(float *x, float *y, float *psi, float v_w, float psi_w, float u_phi[], float u_v[], float Dt)
{   
    float dx, dy, v_g[N+1], chi[N+1], dchi;

    v_g[0] = sqrt( pow(u_v[0]*cos(psi[0])+v_w*cos(psi_w), 2) + pow(u_v[0]*sin(psi[0])+v_w*sin(psi_w), 2) );

    chi[0] = atan2( ( u_v[0]*sin(psi[0]) + v_w*sin(psi_w) ) , ( u_v[0]*cos(psi[0]) + v_w*cos(psi_w) ));
    if (chi[0] < 0) chi[0] = chi[0] + 2*pi;

  //FORWARD EULER METHOD TO INTEGRATE THE STATES OF AGENT (Eq. 02) (Eq. 04)
  for (int k = 0; k < N; k++)
  {
    //x
    dx = v_g[k]*cos(chi[k]);
    x[k+1] = x[k] + dx*Dt;
    //y
    dy = v_g[k]*sin(chi[k]);
    y[k+1] = y[k] + dy*Dt;
    //chi
    dchi = g*tan(u_phi[k])*cos(chi[k] - psi[k])/v_g[k];
    chi[k+1] = chi[k] + dchi*Dt;
    if (chi[k+1]>2*pi) chi[k+1] = chi[k+1]-2*pi;
    //psi
    psi[k+1] = chi[k+1] - asin(v_w/u_v[k+1]*sin(psi_w - chi[k+1]));
    //v_g
    v_g[k+1] = sqrt( pow(u_v[k+1]*cos(psi[k+1])+v_w*cos(psi_w), 2) + pow(u_v[k+1]*sin(psi[k+1])+v_w*sin(psi_w), 2) );
  }
}

__device__ float cuda_cost_function(int index, float controls[], struct agent agents[], struct point2d r[Nb_x][Nb_y], float phi[Nb_x][Nb_y])
{
    for (int k = 0; k < N; k++) agents[index].u_phi[k] = controls[k];
    for (int k = 0; k < N; k++) agents[index].u_v[k] = controls[k+N];
    cuda_forward_euler(agents[index].x, agents[index].y, agents[index].psi, agents[index].v_w, agents[index].psi_w, agents[index].u_phi, agents[index].u_v, Dt_MPC);
    
    //EVALUATE COST FUNCTION
    float F = 0;
    float total_cost = 0.0;
    float sum_phi_y = 0;
    float d[I];

    for (int i = 0; i < I; i++)
    {
        if (i != index) 
        {
          for (int j = 0; j < Nb_x; j++)
          {
            for (int k = 0; k < Nb_y; k++)
            {
              agents[index].B[j][k] = agents[index].B[j][k] || agents[i].b[j][k];
            }
          }
        }
    } 

    //anti colision for future steps. current step doesn't matter
    for (int k = 1; k < N; k++)
    {
      for (int i = 0; i < I; i++) 
      {
        d[i] = sqrt(pow(agents[index].x[k] - agents[i].x[k], 2) + pow(agents[index].y[k] - agents[i].y[k], 2));
        if ((d[i] < r_c) && (i != index)) return 99999999999999999999999999999999999999999999999999999999999999999999.9;
      }
    }

    for (int k = 0; k < (N+1); k++)
    { 
        //Lagrangian term
        float phi_y = 0;
        //check if 50x50 cells are visited
        int x_idx = (int)agents[index].x[k]/res;
        int y_idx = (int)agents[index].y[k]/res;
        for (int i = (x_idx - R/res); i <= (x_idx + R/res); i++)
        {
            for (int j = (y_idx - R/res); j <= (y_idx + R/res); j++)
            {
                //printf("x=%f,y=%f \n",x1[k],y1[k]);
                if (cuda_check_if_grid_is_inside_R(agents[index].x[k],agents[index].y[k],r[i][j]))
                {
                    agents[index].B[i][j] = 1;
                }
            }
        }
        for (int i = 0; i < Nb_x; i++)
        {
            for (int j = 0; j < Nb_y; j++)
            {
                phi_y = phi_y + phi[i][j]*agents[index].B[i][j];
            }
        }
        sum_phi_y = sum_phi_y + phi_y;
    }

    F = cuda_calculate_F(agents[index].x[N],agents[index].y[N],agents[index].B,r,phi); //r_x[Nb], r_y[Nb]: position of center of cells; b[Nb] is the binary of cell visit
    total_cost = F - sum_phi_y; //Eq 13
    return total_cost;
}

__global__ void kernelUpdateParticle(float *positions, float *velocities, 
                                     float *pBests, float *gBest, float r1, 
                                     float r2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i >= NUM_OF_PARTICLES * NUM_OF_DIMENSIONS)
        return;
    
    float rp = r1;
    float rg = r2;

    velocities[i] = OMEGA * velocities[i] + c1 * rp * (pBests[i] - positions[i])
            + c2 * rg * (gBest[i % NUM_OF_DIMENSIONS] - positions[i]);

    // Update posisi particle
    positions[i] += velocities[i];

    if (((i % (N*2)) < N) && (positions[i] < (phi_min))) positions[i] = phi_min;
    if (((i % (N*2)) < N) && (positions[i] > (phi_max))) positions[i] = phi_max;

    if (((i % (N*2)) >= N) && (positions[i] < v_min)) positions[i] = v_min;
    if (((i % (N*2)) >= N) && (positions[i] > v_max)) positions[i] = v_max;
}
//ok
__global__ void kernelUpdatePBest(int index, float *positions, float *pBests, float* gBest, struct agent agents[], struct point2d r[Nb_x][Nb_y], float phi[Nb_x][Nb_y])
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i >= NUM_OF_PARTICLES * NUM_OF_DIMENSIONS || i % NUM_OF_DIMENSIONS != 0)
        return;

    for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
    {
        tempParticle1[j] = positions[i + j];
        tempParticle2[j] = pBests[i + j];
    }

    if (cuda_cost_function(index,tempParticle1,agents,r,phi) < cuda_cost_function(index,tempParticle2,agents,r,phi))
    {
        for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
            pBests[i + k] = positions[i + k];       
    }
}

extern "C" void cuda_pso(int index, float positions[], float velocities[], float pBests[], 
                         float *gBest, struct agent agents[], struct point2d r[Nb_x][Nb_y], float phi[Nb_x][Nb_y])
{
    int size = NUM_OF_PARTICLES * NUM_OF_DIMENSIONS;
    float *devPos;
    float *devVel;
    float *devPBest;
    float *devGBest;
    float temp[NUM_OF_DIMENSIONS];
    // Memory allocation
    hipMalloc((void**)&devPos, sizeof(float) * size);
    hipMalloc((void**)&devVel, sizeof(float) * size);
    hipMalloc((void**)&devPBest, sizeof(float) * size);
    hipMalloc((void**)&devGBest, sizeof(float) * NUM_OF_DIMENSIONS);
    // Thread & Block number
    int threadsNum = 256;
    int blocksNum = NUM_OF_PARTICLES / threadsNum;
    // Copy particle datas from host to device
    hipMemcpy(devPos, positions, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(devVel, velocities, sizeof(float) * size, 
               hipMemcpyHostToDevice);
    hipMemcpy(devPBest, pBests, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(devGBest, gBest, sizeof(float) * NUM_OF_DIMENSIONS, 
               hipMemcpyHostToDevice);

    // PSO main function
    for (int iter = 0; iter < MAX_ITER; iter++)
    {     
        
        // Update position and velocity
        kernelUpdateParticle<<<blocksNum, threadsNum>>>(devPos, devVel, 
                                                        devPBest, devGBest, 
                                                        getRandomClamped(), 
                                                        getRandomClamped());
        // Update pBest
        kernelUpdatePBest<<<blocksNum, threadsNum>>>(index, devPos, devPBest, 
                                                     devGBest,agents,r,phi);
        // Update gBest
        hipMemcpy(pBests, devPBest, 
                   sizeof(float) * NUM_OF_PARTICLES * NUM_OF_DIMENSIONS, 
                   hipMemcpyDeviceToHost);
        for(int i = 0; i < size; i += NUM_OF_DIMENSIONS)
        {
            for(int k = 0; k < NUM_OF_DIMENSIONS; k++)
                temp[k] = pBests[i + k];
        
            if (cost_function(index,temp,agents,r,phi) < cost_function(index, gBest,agents,r,phi))
            {
                for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
                    gBest[k] = temp[k];
            }
        }
        hipMemcpy(devGBest, gBest, sizeof(float) * NUM_OF_DIMENSIONS, 
                   hipMemcpyHostToDevice);
    }
    // Retrieve particle datas from device to host
    hipMemcpy(positions, devPos, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(velocities, devVel, sizeof(float) * size, 
               hipMemcpyDeviceToHost);
    hipMemcpy(pBests, devPBest, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(gBest, devGBest, sizeof(float) * NUM_OF_DIMENSIONS, 
               hipMemcpyDeviceToHost); 
    // cleanup
    hipFree(devPos);
    hipFree(devVel);
    hipFree(devPBest);
    hipFree(devGBest);
}
